#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "cuZC_typeThree.h"
#include "matrix.hpp"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__global__ void type_three(float *data1, float *data2, double *results, int r3, int r2, int r1, int ssimSize, int ssimShift, int yNum) 
{
    //cg::grid_group grid = cg::this_grid();
    float xMin;  
    float xMax; 
    float yMin; 
    float yMax; 
    float xSum; 
    float x2Sum;
    float ySum; 
    float y2Sum;
    float xySum;
    float xdata, ydata, xdata_shflx, ydata_shflx, xdata_shfld, ydata_shfld;

    int i, j;
    int xNum = (warpSize-ssimSize)/ssimShift+1;
    int h=blockIdx.x*yNum*ssimShift;
    if ((h+blockDim.y)>r2) yNum = (r2-h-ssimSize)/ssimShift+1;
    int wstride=xNum*ssimShift;
    double np = ssimSize * ssimSize * ssimSize;
    unsigned mask;
    
    int wsize = (r1-ssimSize+ssimShift)/wstride + ((r1-ssimSize+ssimShift)%wstride?1:0);
    
    //static __shared__ float shared[9*xNum*(yNum*ssimSize+blockDim.y)];
    //static __shared__ float shared[9*26*(2*7+8)];
    extern __shared__ float shared[];

    for (int w=0; w<wsize*wstride; w+=wstride){
        if ((w+blockDim.x)>r1) xNum = (r1-w-ssimSize)/ssimShift+1;
        if (w+threadIdx.x<r1){
            for (int l=0; l<r3; l++){
            //for (int l=0; l<ssimSize; l++){

                xMin = data1[0];  
                xMax = data1[0]; 
                yMin = data2[0]; 
                yMax = data2[0]; 
                xdata= data1[l*r1*r2+(h+threadIdx.y)*r1+(w+threadIdx.x)];
                ydata= data2[l*r1*r2+(h+threadIdx.y)*r1+(w+threadIdx.x)];
                xMin = min(xMin, xdata);
                xMax = max(xMax, xdata);
                yMin = min(yMin, ydata);
                yMax = max(yMax, ydata);
                xSum = xdata; 
                x2Sum = xdata * xdata;
                ySum = ydata; 
                y2Sum = ydata * ydata;
                xySum = xdata * ydata;

                mask = __ballot_sync(FULL_MASK, threadIdx.x < (r1-w));

                for (int offset = 1; offset<ssimSize; offset++) 
                {
                    //xdata_shflx = __shfl_xor_sync(mask, xdata, offset);
                    //ydata_shflx = __shfl_xor_sync(mask, ydata, offset);
                    xdata_shfld = __shfl_down_sync(mask, xdata, offset);
                    ydata_shfld = __shfl_down_sync(mask, ydata, offset);

                    xMin = min(xMin, xdata_shfld);
                    xMax = max(xMax, xdata_shfld);
                    yMin = min(yMin, ydata_shfld);
                    yMax = max(yMax, ydata_shfld);
                    xSum += xdata_shfld;
                    x2Sum += xdata_shfld * xdata_shfld;
                    ySum += ydata_shfld;
                    y2Sum += ydata_shfld * ydata_shfld;
                    xySum += xdata_shfld * ydata_shfld;
                }
                
                if (threadIdx.x%ssimShift==0 && threadIdx.x/ssimShift<xNum){
//if (xMax!=0.0) printf("test%i,%i,%i=%e\n", h,w,l,xMax);
                    shared[(9*yNum*ssimSize+threadIdx.y)*xNum+threadIdx.x/ssimShift] = xMin;
                    shared[(9*yNum*ssimSize+blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = yMin;
                    shared[(9*yNum*ssimSize+2*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = xMax;
                    shared[(9*yNum*ssimSize+3*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = yMax;
                    shared[(9*yNum*ssimSize+4*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = xSum;
                    shared[(9*yNum*ssimSize+5*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = x2Sum;
                    shared[(9*yNum*ssimSize+6*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = ySum;
                    shared[(9*yNum*ssimSize+7*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = y2Sum;
                    shared[(9*yNum*ssimSize+8*blockDim.y+threadIdx.y)*xNum+threadIdx.x/ssimShift] = xySum;

                }
                //if (threadIdx.x==0) printf("test%i,%i=%e\n",l,threadIdx.y,ySum);
                __syncthreads();                  

                if (threadIdx.x<xNum){
                    for (j=0;j<yNum;j++){
                        if (threadIdx.y==j){
                            for (i=j;i<(ssimSize+j);i++) xMin = min(xMin, shared[(9*yNum*ssimSize+i)*xNum+threadIdx.x]);
                            shared[9*(yNum*(l%ssimSize)+j)*xNum+threadIdx.x] = xMin;
                        }else if (threadIdx.y==(j+1)){
                            for (i=j;i<(ssimSize+j);i++) yMin = min(yMin, shared[(9*yNum*ssimSize+blockDim.y+i)*xNum+threadIdx.x]);
                            shared[(9*(yNum*(l%ssimSize)+j)+1)*xNum+threadIdx.x] = yMin;
                        }else if (threadIdx.y==(j+2)){
                            //xMax = shared[(9*yNum*ssimSize+2*blockDim.y)*xNum+threadIdx.x];
                            for (i=j;i<(ssimSize+j);i++) xMax = max(xMax, shared[(9*yNum*ssimSize+2*blockDim.y+i)*xNum+threadIdx.x]);
                            shared[(9*(yNum*(l%ssimSize)+j)+2)*xNum+threadIdx.x] = xMax;
                        }else if (threadIdx.y==(j+3)){
                            for (i=j;i<(ssimSize+j);i++) yMax = max(yMax, shared[(9*yNum*ssimSize+3*blockDim.y+i)*xNum+threadIdx.x]);
                            shared[(9*(yNum*(l%ssimSize)+j)+3)*xNum+threadIdx.x] = yMax;
                        }

                        xSum = 0;
                        if (threadIdx.y<5){
                            for (i=j;i<(ssimSize+j);i++) xSum += shared[(9*yNum*ssimSize+(4+threadIdx.y)*blockDim.y+i)*xNum+threadIdx.x];
                            shared[(9*(yNum*(l%ssimSize)+j)+(4+threadIdx.y))*xNum+threadIdx.x] = xSum;
                        }
                    }
                }
                __syncthreads();                  

                if (l>(ssimSize-2)){
                    if ((l-ssimSize+1)%ssimShift==0){
                        if (threadIdx.x<xNum){
                            if (threadIdx.y<yNum){
                                xMin = shared[(9*(yNum*0+threadIdx.y)+0)*xNum+threadIdx.x];  
                                yMin = shared[(9*(yNum*0+threadIdx.y)+1)*xNum+threadIdx.x]; 
                                xMax = shared[(9*(yNum*0+threadIdx.y)+2)*xNum+threadIdx.x]; 
                                yMax = shared[(9*(yNum*0+threadIdx.y)+3)*xNum+threadIdx.x]; 
                                xSum = shared[(9*(yNum*0+threadIdx.y)+4)*xNum+threadIdx.x]; 
                                x2Sum =shared[(9*(yNum*0+threadIdx.y)+5)*xNum+threadIdx.x];
                                ySum = shared[(9*(yNum*0+threadIdx.y)+6)*xNum+threadIdx.x]; 
                                y2Sum =shared[(9*(yNum*0+threadIdx.y)+7)*xNum+threadIdx.x];
                                xySum =shared[(9*(yNum*0+threadIdx.y)+8)*xNum+threadIdx.x];

                                for (i=1;i<ssimSize;i++) {
                                    xMin = min(xMin, shared[(9*(yNum*i+threadIdx.y)+0)*xNum+threadIdx.x]);
                                    yMin = min(yMin, shared[(9*(yNum*i+threadIdx.y)+1)*xNum+threadIdx.x]);
                                    xMax = max(xMax, shared[(9*(yNum*i+threadIdx.y)+2)*xNum+threadIdx.x]);
                                    yMax = max(yMax, shared[(9*(yNum*i+threadIdx.y)+3)*xNum+threadIdx.x]);
                                    xSum += shared[(9*(yNum*i+threadIdx.y)+4)*xNum+threadIdx.x];
                                    x2Sum += shared[(9*(yNum*i+threadIdx.y)+5)*xNum+threadIdx.x];
                                    ySum += shared[(9*(yNum*i+threadIdx.y)+6)*xNum+threadIdx.x];
                                    y2Sum += shared[(9*(yNum*i+threadIdx.y)+7)*xNum+threadIdx.x];
                                    xySum += shared[(9*(yNum*i+threadIdx.y)+8)*xNum+threadIdx.x];

                                }
                //if (threadIdx.x==0) printf("test%i,%i=%e\n",l,(l-ssimSize+1)/ssimShift*30*yNum+30*threadIdx.y+threadIdx.x,ySum);

                                double xMean=xSum/np;
                                double yMean=ySum/np;
                                double xSigma=sqrt(fabs((x2Sum/np)-(xMean*xMean)));
                                double ySigma=sqrt(fabs((y2Sum/np)-(yMean*yMean)));
                                double xyCov=(xySum/np)-(xMean*yMean);
                                //if (wid==0)printf("ddata%i=%e:%e:%e\n", np, (xySum/np),(xMean*yMean), xyCov);

                                double c1,c2;
                                if(xMax-xMin==0){
                                  c1=K1*K1;
                                  c2=K2*K2;
                                }else{
                                  c1=K1*K1*(xMax-xMin)*(xMax-xMin);
                                  c2=K2*K2*(xMax-xMin)*(xMax-xMin);
                                }
                                double c3=c2/2;
                                  
                                double luminance=(2*xMean*yMean+c1)/(xMean*xMean+yMean*yMean+c1);
                                double contrast=(2*xSigma*ySigma+c2)/(xSigma*xSigma+ySigma*ySigma+c2);
                                double structure=(xyCov+c3)/(xSigma*ySigma+c3);
                                double ssim=luminance*contrast*structure;

                                results[(l-ssimSize+1)/ssimShift*r1*r2+r1*(h+threadIdx.y)+(w+threadIdx.x)] = ssim;
                            }
                        }
                        __syncthreads();                  
                    }
                }
            }

        }

    }
    //cg::sync(grid);
}

__global__ void gridR_typeThree(double *results, int size) 
{
    int tidx = threadIdx.x;
    double data = results[tidx];

    for (int i=(tidx+blockDim.x); i<size; i+=blockDim.x)
        data += results[i];
    __syncthreads();                  

    for (int offset = warpSize/2; offset > 0; offset /= 2) 
        data += __shfl_down_sync(FULL_MASK, data, offset);

    if (tidx==0) results[0] = data;
        
}
