#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "cuZC_typeThree.h"
#include "matrix.hpp"

__global__ void type_three(float *data1, float *data2, double *results, int r3, int r2, int r1, int ssimSize, int ssimShift) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;
    float xMin;  
    float xMax; 
    float yMin; 
    float yMax; 
    float xSum; 
    float x2Sum;
    float ySum; 
    float y2Sum;
    float xySum;
    float xdata, ydata, xdata_shflx, ydata_shflx, xdata_shfld, ydata_shfld;

    int i, j;
    int xNum = (warpSize-ssimSize)/ssimShift+1;
    int yNum = (blockDim.y-ssimSize)/ssimShift+1;
    int h=bid*yNum*ssimShift;
    int wstride=xNum*ssimShift;
    double np = ssimSize * ssimSize * ssimSize;
    
    int wsize = (r1-ssimSize+ssimShift)/wstride + ((r1-ssimSize+ssimShift)%wstride?1:0);
    
    //static __shared__ float shared[9*xNum*(yNum*ssimSize+blockDim.y)];
    static __shared__ float shared[9*26*(2*7+8)];

    //for (int w=0; w<wsize*wstride; w+=wstride){
    int w=0;
        for (int l=0; l<r3; l++){
            xMin = data1[0];  
            xMax = data1[0]; 
            yMin = data2[0]; 
            yMax = data2[0]; 
            xdata= data1[l*r1*r2+(h+tidy)*r1+(w+tidx)];
            ydata= data2[l*r1*r2+(h+tidy)*r1+(w+tidx)];
            xMin = min(xMin, xdata);
            xMax = max(xMax, xdata);
            yMin = min(yMin, ydata);
            yMax = max(yMax, ydata);
            xSum = xdata; 
            x2Sum = xdata * xdata;
            ySum = ydata; 
            y2Sum = ydata * ydata;
            xySum = xdata * ydata;


            for (int offset = 1; offset<ssimSize; offset++) 
            {
                xdata_shflx = __shfl_xor_sync(FULL_MASK, xdata, offset);
                ydata_shflx = __shfl_xor_sync(FULL_MASK, ydata, offset);
                xdata_shfld = __shfl_down_sync(FULL_MASK, xdata, offset);
                ydata_shfld = __shfl_down_sync(FULL_MASK, ydata, offset);

                xMin = min(xMin, xdata_shflx);
                xMax = max(xMax, xdata_shflx);
                yMin = min(yMin, ydata_shflx);
                yMax = max(yMax, ydata_shflx);
                xSum += xdata_shfld;
                x2Sum += xdata_shfld * xdata_shfld;
                ySum += ydata_shfld;
                y2Sum += ydata_shfld * ydata_shfld;
                xySum += xdata_shfld * ydata_shfld;
            }
            
            if (tidx<xNum){
                shared[(9*yNum*ssimSize+tidy)*xNum+tidx] = xMin;
                shared[(9*yNum*ssimSize+blockDim.y+tidy)*xNum+tidx] = yMin;
                shared[(9*yNum*ssimSize+2*blockDim.y+tidy)*xNum+tidx] = xMax;
                shared[(9*yNum*ssimSize+3*blockDim.y+tidy)*xNum+tidx] = yMax;
                shared[(9*yNum*ssimSize+4*blockDim.y+tidy)*xNum+tidx] = xSum;
                shared[(9*yNum*ssimSize+5*blockDim.y+tidy)*xNum+tidx] = x2Sum;
                shared[(9*yNum*ssimSize+6*blockDim.y+tidy)*xNum+tidx] = ySum;
                shared[(9*yNum*ssimSize+7*blockDim.y+tidy)*xNum+tidx] = y2Sum;
                shared[(9*yNum*ssimSize+8*blockDim.y+tidy)*xNum+tidx] = xySum;

            }
            //if (tidx==0) printf("test%i,%i=%e\n",l,tidy,ySum);
            __syncthreads();                  

            if (tidx<xNum){
                for (j=0;j<yNum;j++){
                    if (tidy==j){
                        for (i=j;i<(ssimSize+j);i++) xMin = min(xMin, shared[(9*yNum*ssimSize+i)*xNum+tidx]);
                        shared[9*(yNum*(l%ssimSize)+j)*xNum+tidx] = xMin;
                    }else if (tidy==(j+1)){
                        for (i=j;i<(ssimSize+j);i++) yMin = min(yMin, shared[(9*yNum*ssimSize+blockDim.y+i)*xNum+tidx]);
                        shared[(9*(yNum*(l%ssimSize)+j)+1)*xNum+tidx] = yMin;
                    }else if (tidy==(j+2)){
                        for (i=j;i<(ssimSize+j);i++) xMax = max(xMax, shared[(9*yNum*ssimSize+2*blockDim.y+i)*xNum+tidx]);
                        shared[(9*(yNum*(l%ssimSize)+j)+2)*xNum+tidx] = xMax;
                    }else if (tidy==(j+3)){
                        for (i=j;i<(ssimSize+j);i++) yMax = max(yMax, shared[(9*yNum*ssimSize+3*blockDim.y+i)*xNum+tidx]);
                        shared[(9*(yNum*(l%ssimSize)+j)+3)*xNum+tidx] = yMax;
                    }

                    xSum = 0;
                    if (tidy<5){
                        for (i=j;i<(ssimSize+j);i++) xSum += shared[(9*yNum*ssimSize+(4+tidy)*blockDim.y+i)*xNum+tidx];
                        shared[(9*(yNum*(l%ssimSize)+j)+(4+tidy))*xNum+tidx] = xSum;
                    }
                }
            }
            __syncthreads();                  

            if (l>(ssimSize-2)){
                if ((l-ssimSize+1)%ssimShift==0){
                    if (tidx<xNum){
                        if (tidy<yNum){
                            xMin = shared[(9*(yNum*0+tidy)+0)*xNum+tidx];  
                            yMin = shared[(9*(yNum*0+tidy)+1)*xNum+tidx]; 
                            xMax = shared[(9*(yNum*0+tidy)+2)*xNum+tidx]; 
                            yMax = shared[(9*(yNum*0+tidy)+3)*xNum+tidx]; 
                            xSum = shared[(9*(yNum*0+tidy)+4)*xNum+tidx]; 
                            x2Sum =shared[(9*(yNum*0+tidy)+5)*xNum+tidx];
                            ySum = shared[(9*(yNum*0+tidy)+6)*xNum+tidx]; 
                            y2Sum =shared[(9*(yNum*0+tidy)+7)*xNum+tidx];
                            xySum =shared[(9*(yNum*0+tidy)+8)*xNum+tidx];

                            for (i=1;i<ssimSize;i++) {
                                xMin = min(xMin, shared[(9*(yNum*i+tidy)+0)*xNum+tidx]);
                                yMin = min(yMin, shared[(9*(yNum*i+tidy)+1)*xNum+tidx]);
                                xMax = max(xMax, shared[(9*(yNum*i+tidy)+2)*xNum+tidx]);
                                yMax = max(yMax, shared[(9*(yNum*i+tidy)+3)*xNum+tidx]);
                                xSum += shared[(9*(yNum*i+tidy)+4)*xNum+tidx];
                                x2Sum += shared[(9*(yNum*i+tidy)+5)*xNum+tidx];
                                ySum += shared[(9*(yNum*i+tidy)+6)*xNum+tidx];
                                y2Sum += shared[(9*(yNum*i+tidy)+7)*xNum+tidx];
                                xySum += shared[(9*(yNum*i+tidy)+8)*xNum+tidx];

                            }
            if (tidx==0) printf("test%i,%i=%e\n",l,(l-ssimSize+1)/ssimShift*30*yNum+30*tidy+tidx,ySum);

                            double xMean=xSum/np;
                            double yMean=ySum/np;
                            double xSigma=sqrt(fabs((x2Sum/np)-(xMean*xMean)));
                            double ySigma=sqrt(fabs((y2Sum/np)-(yMean*yMean)));
                            double xyCov=(xySum/np)-(xMean*yMean);
                            //if (wid==0)printf("ddata%i=%e:%e:%e\n", np, (xySum/np),(xMean*yMean), xyCov);

                            double c1,c2;
                            if(xMax-xMin==0){
                              c1=K1*K1;
                              c2=K2*K2;
                            }else{
                              c1=K1*K1*(xMax-xMin)*(xMax-xMin);
                              c2=K2*K2*(xMax-xMin)*(xMax-xMin);
                            }
                            double c3=c2/2;
                              
                            double luminance=(2*xMean*yMean+c1)/(xMean*xMean+yMean*yMean+c1);
                            double contrast=(2*xSigma*ySigma+c2)/(xSigma*xSigma+ySigma*ySigma+c2);
                            double structure=(xyCov+c3)/(xSigma*ySigma+c3);
                            double ssim=luminance*contrast*structure;

                            results[(l-ssimSize+1)/ssimShift*30*yNum+30*tidy+tidx] = ySum;
                        }
                    }
                    __syncthreads();                  
                }
            }
        }

    //}
}
