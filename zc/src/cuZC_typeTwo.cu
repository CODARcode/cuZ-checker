#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "cuZC_typeTwo.h"
#include "matrix.hpp"

__global__ void type_two(float *data, float *der, float *autocor, int r3, int r2, int r1, float avg, size_t order) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;

    float base, sum;
    int i, j;
    int h=blockIdx.x*(16-order*2);
    double dx, dy, dz;
    
    int wsize = (r2-order*2)/(16-order*2) + ((r2-order*2)%(16-order*2)?1:0);
    int lsize = (r1-order*2)/(16-order*2) + ((r1-order*2)%(16-order*2)?1:0);
    
    //static __shared__ float shared[16*16*16];
    extern __shared__ float shared[];
    float *bdata = shared;
    float *cor = &shared[blockDim.x * blockDim.y * 16];
    unsigned mask;

    for (int w=0; w<wsize*(16-order*2); w+=(16-order*2)){
        for (int l=0; l<lsize*(16-order*2); l+=(16-order*2)){
            for (i=0; i<16; i++){
                if ((h+i)<r3 && (l+tidx)<r1 && (w+tidy)<r2){
                    bdata[i*16*16+tidy*16+tidx] = data[(h+i)*r1*r2+(w+tidy)*r1+(l+tidx)];
                }
                    
            }
            __syncthreads();                  

            for (i=0; i<(16-order*2); i++){
                if (tidx<(16-order*2) && tidy<(16-order*2)){
                    if ((h+i)<(r3-order*2) && (l+tidx)<(r1-order*2) && (w+tidy)<(r2-order*2)){
                        base = bdata[(i+order)*16*16+(tidy+order)*16+tidx];
                        dx = (bdata[(i+order)*16*16+(tidy+order)*16+tidx+order*2] - base)/2;
                        base = bdata[(i+order)*16*16+tidy*16+tidx+order];
                        dy = (bdata[(i+order)*16*16+(tidy+order*2)*16+tidx+order] - base)/2;
                        base = bdata[i*16*16+(tidy+order)*16+tidx+order];
                        dz = (bdata[(i+order*2)*16*16+(tidy+order)*16+tidx+order] - base)/2;
                        //if (bid==0)printf("index=%i,%e,%i,%i\n",i+2,dz,h+tidy,l+tidx);
                        //if (Data!=0.0) printf("ddata%i,%i,%i,%i,%i,%i=%e\n",w,l,bid,i,tidx,tidy,sqrt(dx*dx+dy*dy+dz*dz));
                        der[(h+i)*(r1-order*2)*(r2-order*2)+(w+tidy)*(r1-order*2)+(l+tidx)] = sqrt(dx*dx+dy*dy+dz*dz);
                        //if (der[(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx)]!=0.0) printf("ddata%i=%e\n",(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx),der[(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx)]);

                        mask = __ballot_sync(FULL_MASK, 1);
                        base = bdata[i*16*16+tidy*16+tidx];

                        for (j=1; j<=order*2; j++){
                            sum = (bdata[(i+j)*16*16+(tidy+j)*16+tidx+j]-avg) * (base-avg);

                            for (int offset = warpSize/2; offset > 0; offset /= 2) 
                                sum += __shfl_down_sync(mask, sum, offset);

                            if (tidx==0) cor[blockDim.y*(j-1)+tidy] = sum;
                        }
                    }
                }
                __syncthreads();                  

                if (tidy<order*2){
                    if (tidx < (16-order*2) && (w+tidx)<(r2-order*2))
                    {
                        sum = cor[blockDim.y*tidy+tidx];
                        mask = __ballot_sync(FULL_MASK, 1);
                    } else sum = 0;
                    for (int offset = warpSize/2; offset > 0; offset /= 2) 
                        sum += __shfl_down_sync(mask, sum, offset);

                    if (tidx==0) autocor[gridDim.x*tidy+blockIdx.x] += sum;
                }
                __syncthreads();                  
            }
        }
    }
}
