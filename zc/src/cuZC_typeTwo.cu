#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "cuZC_typeTwo.h"
#include "matrix.hpp"

__global__ void type_two(float *data, float *der, int r3, int r2, int r1, size_t order) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float Data;
    int i, j;
    int h=bid*(16-order*2);
    double dx, dy, dz;
    
    int wsize = (r2-order*2)/(16-order*2) + ((r2-order*2)%(16-order*2)?1:0);
    int lsize = (r1-order*2)/(16-order*2) + ((r1-order*2)%(16-order*2)?1:0);
    
    static __shared__ float shared[16*16*16];

    for (int w=0; w<wsize*(16-order*2); w+=(16-order*2)){
        for (int l=0; l<lsize*(16-order*2); l+=(16-order*2)){
            for (i=0; i<16; i++){
                if ((h+i)<r3 && (l+tidx)<r1 && (w+tidy)<r2){
                    shared[i*16*16+tidy*16+tidx] = data[(h+i)*r1*r2+(w+tidy)*r1+(l+tidx)];
                }
                    
            }
            __syncthreads();                  

            for (i=0; i<(16-order*2); i++){
                if (tidx<(16-order*2) && tidy<(16-order*2)){
                    if ((h+i)<(r3-order*2) && (l+tidx)<(r1-order*2) && (w+tidy)<(r2-order*2)){
                        Data = shared[i*16*16+tidy*16+tidx];
                        dx = (shared[i*16*16+tidy*16+tidx+order*2] - Data)/2;
                        dy = (shared[i*16*16+(tidy+order*2)*16+tidx] - Data)/2;
                        dz = (shared[(i+order*2)*16*16+tidy*16+tidx] - Data)/2;
                        //if (bid==0)printf("index=%i,%e,%i,%i\n",i+2,dz,h+tidy,l+tidx);
                        //if (Data!=0.0) printf("ddata%i,%i,%i,%i,%i,%i=%e\n",w,l,bid,i,tidx,tidy,sqrt(dx*dx+dy*dy+dz*dz));
                        der[(h+i)*(r1-order*2)*(r2-order*2)+(w+tidy)*(r1-order*2)+(l+tidx)] = sqrt(dx*dx+dy*dy+dz*dz);
                        //der[(h+i)*(r1-order*2)*(r2-order*2)+(w+tidy)*(r1-order*2)+(l+tidx)] = Data;
                        //if (der[(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx)]!=0.0) printf("ddata%i=%e\n",(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx),der[(w+i)*(r1-order*2)*(r2-order*2)+(h+tidy)*(r1-order*2)+(l+tidx)]);
                    
                    }
                }
            }
            __syncthreads();                  
        }
    }

}

