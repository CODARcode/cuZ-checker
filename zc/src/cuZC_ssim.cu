#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "matrix.hpp"

__global__ void hello(char *a, int *b) 
{
    a[threadIdx.x] += b[threadIdx.x];
}

__device__
double ssim_winComp(float xMin, float xMax, float yMin, float yMax, float xSum, float x2Sum, float ySum, float y2Sum, float xySum, double np) {

    static __shared__ float shared[32*9]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;


    for (int offset = warpSize/2; offset > 0; offset /= 2) 
    {
        xMin = min(xMin, __shfl_xor_sync(FULL_MASK, xMin, offset));
        xMax = max(xMax, __shfl_xor_sync(FULL_MASK, xMax, offset));
        yMin = min(yMin, __shfl_xor_sync(FULL_MASK, yMin, offset));
        yMax = max(yMax, __shfl_xor_sync(FULL_MASK, yMax, offset));
        xSum += __shfl_down_sync(FULL_MASK, xSum, offset);
        x2Sum += __shfl_down_sync(FULL_MASK, x2Sum, offset);
        ySum += __shfl_down_sync(FULL_MASK, ySum, offset);
        y2Sum += __shfl_down_sync(FULL_MASK, y2Sum, offset);
        xySum += __shfl_down_sync(FULL_MASK, xySum, offset);
    }

    if (lane==0){
        shared[wid] = xMin;
        shared[32+wid] = xMax;
        shared[32*2+wid] = yMin;
        shared[32*3+wid] = yMax;
        shared[32*4+wid] = xSum;
        shared[32*5+wid] = x2Sum;
        shared[32*6+wid] = ySum;
        shared[32*7+wid] = y2Sum;
        shared[32*8+wid] = xySum;
        //printf("shared%i=%e\n", 32*6+wid, shared[32*6+wid]);
    }

    __syncthreads();                  

    if (threadIdx.x < blockDim.x / warpSize){
        xMin = shared[lane];
        xMax = shared[32+lane];
        yMin = shared[32*2+lane];
        yMax = shared[32*3+lane];
        xSum = shared[32*4+lane];
        x2Sum = shared[32*5+lane];
        ySum = shared[32*6+lane];
        y2Sum = shared[32*7+lane];
        xySum = shared[32*8+lane];
    }else{
        xMin = 0;  
        xMax = 0; 
        yMin = 0; 
        yMax = 0; 
        xSum = 0; 
        x2Sum = 0;
        ySum = 0; 
        y2Sum = 0;
        xySum = 0;
    }
    //if (wid==0)printf("ddata%i=%e:%e\n", 32*6+lane, shared[32*6+lane], ySum);

    if (wid==0){
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            xMin = min(xMin, __shfl_xor_sync(FULL_MASK, xMin, offset));
            xMax = max(xMax, __shfl_xor_sync(FULL_MASK, xMax, offset));
            yMin = min(yMin, __shfl_xor_sync(FULL_MASK, yMin, offset));
            yMax = max(yMax, __shfl_xor_sync(FULL_MASK, yMax, offset));
            xSum += __shfl_down_sync(FULL_MASK, xSum, offset);
            x2Sum += __shfl_down_sync(FULL_MASK, x2Sum, offset);
            ySum += __shfl_down_sync(FULL_MASK, ySum, offset);
            y2Sum += __shfl_down_sync(FULL_MASK, y2Sum, offset);
            xySum += __shfl_down_sync(FULL_MASK, xySum, offset);
        }
    }
    double xMean=xSum/np;
    double yMean=ySum/np;
    double xSigma=sqrt(fabs((x2Sum/np)-(xMean*xMean)));
    double ySigma=sqrt(fabs((y2Sum/np)-(yMean*yMean)));
    double xyCov=(xySum/np)-(xMean*yMean);
    //if (wid==0)printf("ddata%i=%e:%e:%e\n", np, (xySum/np),(xMean*yMean), xyCov);

    double c1,c2;
    if(xMax-xMin==0){
      c1=K1*K1;
      c2=K2*K2;
    }else{
      c1=K1*K1*(xMax-xMin)*(xMax-xMin);
      c2=K2*K2*(xMax-xMin)*(xMax-xMin);
    }
    double c3=c2/2;
      
    double luminance=(2*xMean*yMean+c1)/(xMean*xMean+yMean*yMean+c1);
    double contrast=(2*xSigma*ySigma+c2)/(xSigma*xSigma+ySigma*ySigma+c2);
    double structure=(xyCov+c3)/(xSigma*ySigma+c3);
    double ssim=luminance*contrast*structure;
    //if (wid==0)printf("ddata%i=%e:%e:%e:%e:%e:%e\n", np,xMean,yMean,xSigma,ySigma,xyCov,ssim);
    return ssim;
    return ySum;
}

__global__ void ssim(float *data1, float *data2, double *results, int r3, int r2, int r1, int ssimSize, int ssimShift) 
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    float xMin;  
    float xMax; 
    float yMin; 
    float yMax; 
    float xSum; 
    float x2Sum;
    float ySum; 
    float y2Sum;
    float xySum;
    int i;
    int np = ssimSize * ssimSize * ssimSize;
    double result = 0;

    int Offsetx;
    int Offsety;
    int Offsetz = bid * ssimShift;
    //int Offsetz = 0;
    int x=0;

    for (Offsetx=0; Offsetx+ssimSize<=r3; Offsetx+=ssimShift){
        for (Offsety=0; Offsety+ssimSize<=r2; Offsety+=ssimShift){
    //for (Offsetx=0; Offsetx+ssimSize<=ssimSize; Offsetx+=ssimShift){
    //    for (Offsety=0; Offsety+ssimSize<=ssimSize; Offsety+=ssimShift){
    //for (Offsetx=36; Offsetx<=36; Offsetx+=ssimShift){
    //    for (Offsety=330; Offsety<=331; Offsety+=ssimShift){
            xMin = data1[0];  
            xMax = data1[0]; 
            yMin = data2[0]; 
            yMax = data2[0]; 
            xSum = 0; 
            x2Sum = 0;
            ySum = 0; 
            y2Sum = 0;
            xySum = 0;

            for (i=tid; i<ssimSize*ssimSize*ssimSize; i+=blockDim.x)
            {
                int Winx = i / (ssimSize*ssimSize);
                int Winy = i % (ssimSize*ssimSize) / ssimSize;
                int Winz = i % (ssimSize*ssimSize) % ssimSize;
                int index = (Offsetx + Winx) * r1 * r2 + (Offsety + Winy) * r2 + (Offsetz + Winz);
                float xdata = data1[index];
                float ydata = data2[index];
                //if (i<blockDim.x)
                //{
                //    xMin = xdata;  
                //    xMax = xdata; 
                //    yMin = ydata; 
                //    yMax = ydata; 
                //}
                xMin = min(xMin, xdata);
                xMax = max(xMax, xdata);
                yMin = min(yMin, ydata);
                yMax = max(yMax, ydata);
                xSum += xdata;
                x2Sum += xdata * xdata;
                ySum += ydata;
                y2Sum += ydata * ydata;
                xySum += xdata * ydata;
            //printf("ydata%i=%e\n",index, xdata);

            }
            //printf("ydata=%e\n", ySum);
            //if (tid == 0) results[bid] = blockReduceSum(val);
            result += ssim_winComp(xMin, xMax, yMin, yMax, xSum, x2Sum, ySum, y2Sum, xySum, np);
            //results[x++] = ssim_winComp(xMin, xMax, yMin, yMax, xSum, x2Sum, ySum, y2Sum, xySum, np);
            //results[tid] = blockReduceSum(val);
            __syncthreads();                  

        }
    }
if (tid == 0)printf("ydata%i,%i=%e\n",Offsetx,Offsety, result);
    if (tid==0) results[bid] = result;
    
}
