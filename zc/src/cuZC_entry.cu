#include "cuZC_entry.h"
#include "timingGPU.h"

TimingGPU timer_GPU;
 
double cu_SSIM_3d_windowed(int windowSize0, int windowSize1, int windowSize2, int windowShift0, int windowShift1, int windowShift2)
{
    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
     
    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);
     
    printf("%s", a);
     
    hipMalloc( (void**)&ad, csize  ); 
    hipMalloc( (void**)&bd, isize  ); 
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice  ); 
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice  ); 

    dim3 dimBlock( blocksize, 1  );
    dim3 dimGrid( 1, 1  );
    //hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost  ); 
    hipFree( ad  );
    hipFree( bd  );

    printf("%s\n", a);
}

int cu_SSIM(float *data1, float *data2, size_t r3, size_t r2, size_t r1, int ssimSize, int ssimShift)
{
    float data[246];
    for (int i=0; i<246; i++){
        data[i] = 1;
    }
    int blksize = (r1 - ssimSize) / ssimShift + 1;
    int xsize = ((r2 - ssimSize) / ssimShift + 1)*((r3 - ssimSize) / ssimShift + 1);

    double results[blksize] = { 0 };
    //printf("test=%f, %f\n", data[32], results[32]);

    float *ddata1, *ddata2;
    double *dresults;
    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}

    const int csize = r3 * r2 * r1 * sizeof(float);
    const int isize = blksize * sizeof(double);

    hipMalloc((void**)&ddata1,   csize); 
    hipMalloc((void**)&ddata2,   csize); 
    hipMalloc((void**)&dresults, isize); 
    hipMemcpy(ddata1,   data1,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(ddata2,   data2,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(dresults, results, isize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    dim3 dimBlock(64, 1);
    dim3 dimGrid(blksize, 1);
    ssim<<<dimGrid, dimBlock>>>(ddata1, ddata2, dresults, r3, r2, r1, ssimSize, ssimShift);
    hipMemcpy(results, dresults, isize, hipMemcpyDeviceToHost); 
    double x=0;
    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    for (int i=0; i<blksize; i++){
        x += results[i];
        printf("results%i=%e\n",i,x);

    }

    hipFree(ddata1);
    hipFree(ddata2);
    hipFree(dresults);

    return 0;
}
