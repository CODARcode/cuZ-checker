#include "cuZC_entry.h"
#include "timingGPU.h"

TimingGPU timer_GPU;
 
double cu_SSIM_3d_windowed(int windowSize0, int windowSize1, int windowSize2, int windowShift0, int windowShift1, int windowShift2)
{
    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
     
    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);
     
    printf("%s", a);
     
    hipMalloc( (void**)&ad, csize  ); 
    hipMalloc( (void**)&bd, isize  ); 
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice  ); 
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice  ); 

    dim3 dimBlock( blocksize, 1  );
    dim3 dimGrid( 1, 1  );
    //hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost  ); 
    hipFree( ad  );
    hipFree( bd  );

    printf("%s\n", a);
}

int cu_SSIM(float *data1, float *data2, size_t r3, size_t r2, size_t r1, int ssimSize, int ssimShift)
{
    float data[246];
    for (int i=0; i<246; i++){
        data[i] = 1;
    }
    int blksize = (r1 - ssimSize) / ssimShift + 1;
    int xsize = ((r2 - ssimSize) / ssimShift + 1)*((r3 - ssimSize) / ssimShift + 1);

    double results[blksize] = { 0 };
    //printf("test=%f, %f\n", data[32], results[32]);

    float *ddata1, *ddata2;
    double *dresults;
    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}

    const int csize = r3 * r2 * r1 * sizeof(float);
    const int isize = blksize * sizeof(double);

    hipMalloc((void**)&ddata1,   csize); 
    hipMalloc((void**)&ddata2,   csize); 
    hipMalloc((void**)&dresults, isize); 
    hipMemcpy(ddata1,   data1,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(ddata2,   data2,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(dresults, results, isize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    dim3 dimBlock(64, 1);
    dim3 dimGrid(blksize, 1);
    ssim<<<dimGrid, dimBlock>>>(ddata1, ddata2, dresults, r3, r2, r1, ssimSize, ssimShift);
    hipMemcpy(results, dresults, isize, hipMemcpyDeviceToHost); 
    double x=0;
    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    for (int i=0; i<blksize; i++){
        x += results[i];
        printf("results%i=%e\n",i,x);

    }

    hipFree(ddata1);
    hipFree(ddata2);
    hipFree(dresults);

    return 0;
}

double *cu_typeOne(float *ddata1, float *ddata2, double *ddiff, double *absErrPDF, double *results, size_t r3, size_t r2, size_t r1, size_t ne){

    //float *ddata1, *ddata2;
    double *dabsErrPDF, *dresults;
    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}

    const int dsize = ne * sizeof(double);
    const int rsize = r3 * 10 * sizeof(double);

    hipMalloc((void**)&dabsErrPDF, dsize); 
    hipMalloc((void**)&dresults, rsize); 
    hipMemcpy(dresults, results, rsize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    dim3 dimBlock(32, 8);
    dim3 dimGrid(r3, 1);
    type_one<<<dimGrid, dimBlock>>>(ddata1, ddata2, ddiff, dresults, r3, r2, r1, ne);

    dim3 dimBlock2(32, 10);
    gridReduction<<<1, dimBlock2>>>(dresults, r3);

    hipMemcpy(results, dresults, rsize, hipMemcpyDeviceToHost); 
    double x=0;
    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    //for (int i=0; i<r3; i++){
    //    x += results[i];
    //    printf("results%i=%e\n",i,x);

    //}

    hipFree(dabsErrPDF);
    hipFree(dresults);

    return results;
}

float *cu_typeTwo(float *ddata, float *der, size_t r3, size_t r2, size_t r1, size_t order){

    float *dder;
    const int dsize = (r3-order*2) * (r2-order*2) * (r1-order*2) * sizeof(float);

    hipMalloc((void**)&dder, dsize); 
    hipMemcpy(dder, der, dsize, hipMemcpyHostToDevice); 

    int blksize = (r3-order*2)/(16-order*2)+((r3-order*2)%(16-order*2)?1:0);
    timer_GPU.StartCounter();
    dim3 dimBlock(16, 16);
    dim3 dimGrid(blksize, 1);
    type_two<<<dimGrid, dimBlock>>>(ddata, dder, r3, r2, r1, order);

    hipMemcpy(der, dder, dsize, hipMemcpyDeviceToHost); 

    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    //for (int i=0;i<(r3-4)*(r2-4)*(r1-4);i++){
    //    if (der[i]!=0.0) printf("ddata%i=%e\n",i,der[i]);
    //}

    hipFree(dder);

    return der;
}
