#include "cuZC_entry.h"
#include "timingGPU.h"

TimingGPU timer_GPU;
 
double cu_SSIM_3d_windowed(int windowSize0, int windowSize1, int windowSize2, int windowShift0, int windowShift1, int windowShift2)
{
    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
     
    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);
     
    printf("%s", a);
     
    hipMalloc( (void**)&ad, csize  ); 
    hipMalloc( (void**)&bd, isize  ); 
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice  ); 
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice  ); 

    dim3 dimBlock( blocksize, 1  );
    dim3 dimGrid( 1, 1  );
    //hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost  ); 
    hipFree( ad  );
    hipFree( bd  );

    printf("%s\n", a);
}

int cu_SSIM(float *data1, float *data2, size_t r3, size_t r2, size_t r1, int ssimSize, int ssimShift)
{
    float data[246];
    for (int i=0; i<246; i++){
        data[i] = 1;
    }
    int blksize = (r1 - ssimSize) / ssimShift + 1;
    int xsize = ((r2 - ssimSize) / ssimShift + 1)*((r3 - ssimSize) / ssimShift + 1);

    double results[blksize] = { 0 };
    //printf("test=%f, %f\n", data[32], results[32]);

    float *ddata1, *ddata2;
    double *dresults;
    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}

    const int csize = r3 * r2 * r1 * sizeof(float);
    const int isize = blksize * sizeof(double);

    hipMalloc((void**)&ddata1,   csize); 
    hipMalloc((void**)&ddata2,   csize); 
    hipMalloc((void**)&dresults, isize); 
    hipMemcpy(ddata1,   data1,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(ddata2,   data2,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(dresults, results, isize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    dim3 dimBlock(64, 1);
    dim3 dimGrid(blksize, 1);
    ssim<<<dimGrid, dimBlock>>>(ddata1, ddata2, dresults, r3, r2, r1, ssimSize, ssimShift);
    hipMemcpy(results, dresults, isize, hipMemcpyDeviceToHost); 
    double x=0;
    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    for (int i=0; i<blksize; i++){
        x += results[i];
        printf("results%i=%e\n",i,x);

    }

    hipFree(ddata1);
    hipFree(ddata2);
    hipFree(dresults);

    return 0;
}

double *cu_typeOne(float *ddata1, float *ddata2, double *ddiff, double *absErrPDF, double *results, size_t r3, size_t r2, size_t r1, size_t ne){

    //float *ddata1, *ddata2;
    double *dabsErrPDF, *dresults;
    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}

    const int dsize = ne * sizeof(double);
    const int rsize = r3 * 10 * sizeof(double);

    hipMalloc((void**)&dabsErrPDF, dsize); 
    hipMalloc((void**)&dresults, rsize); 
    hipMemcpy(dresults, results, rsize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    void *kernelArgs[] = {
        (void *)&ddata1, (void *)&ddata2, (void *)&ddiff, (void *)&dresults, 
        (void *)&r3, (void *)&r2, (void *)&r1, (void *)&ne,
    };
    dim3 dimBlock(32, 10);
    dim3 dimGrid(r3, 1);
    const int sMemsize = 10 * dimBlock.y * sizeof(double);
    printf("test:%i,%i\n",sMemsize, dimBlock.y);
    //type_one<<<dimGrid, dimBlock>>>(ddata1, ddata2, ddiff, dresults, r3, r2, r1, ne);
    checkCudaErrors(hipLaunchCooperativeKernel((void*)type_one,
                                                dimGrid, dimBlock, kernelArgs, sMemsize));

    //dim3 dimBlock2(32, 10);
    //gridReduction<<<1, dimBlock2>>>(dresults, r3);

    hipMemcpy(results, dresults, rsize, hipMemcpyDeviceToHost); 
    double x=0;
    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    //for (int i=0; i<r3; i++){
    //    x += results[i];
    //    printf("results%i=%e\n",i,x);

    //}

    hipFree(dabsErrPDF);
    hipFree(dresults);

    return results;
}

float *cu_typeTwo(float *ddata, float *der, size_t r3, size_t r2, size_t r1, double avg, size_t order){

    float *dder, *autocor, *dautocor;
    const int dsize = (r3-order*2) * (r2-order*2) * (r1-order*2) * sizeof(float);
    int gridsize = (r3-order*2)/(16-order*2)+((r3-order*2)%(16-order*2)?1:0);
    int corsize = gridsize * order * 2 * sizeof(float);

    hipMalloc((void**)&dder, dsize); 
    hipMemcpy(dder, der, dsize, hipMemcpyHostToDevice); 
    autocor = (float*)malloc(corsize);
    memset(autocor, 0, corsize);
    hipMalloc((void**)&dautocor, corsize); 
    hipMemcpy(dautocor, autocor, corsize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    dim3 dimBlock(16, 16);
    dim3 dimGrid(gridsize, 1);
    const int sMemsize = (16 * dimBlock.x * dimBlock.y + dimBlock.y * order * 2) * sizeof(double);
    //type_two<<<dimGrid, dimBlock>>>(ddata, dder, r3, r2, r1, order);
    void *kernelArgs[] = {
        (void *)&ddata, (void *)&dder, (void *)&dautocor,
        (void *)&r3, (void *)&r2, (void *)&r1, (void *)&avg, (void *)&order,
    };
    checkCudaErrors(hipLaunchCooperativeKernel((void*)type_two,
                                                dimGrid, dimBlock, kernelArgs, sMemsize));

    hipMemcpy(der, dder, dsize, hipMemcpyDeviceToHost); 
    hipMemcpy(autocor, dautocor, corsize, hipMemcpyDeviceToHost); 

    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    //for (int i=0;i<(r3-4)*(r2-4)*(r1-4);i++){
    //    if (der[i]!=0.0) printf("ddata%i=%e\n",i,der[i]);
    //}

    hipFree(dder);
    hipFree(dautocor);

    return autocor;
}

double cu_typeThree(float *data1, float *data2, int r3, int r2, int r1, int ssimSize, int ssimShift) 
{

    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}
    float *ddata1, *ddata2;
    double *dresults;
	double *results = (double*)malloc(r2*r1*((r3-ssimSize+1)/ssimShift)*sizeof(double));
	//double *results = (double*)malloc(r2*r1*sizeof(double));
    memset(results, 0, r2*r1*((r3-ssimSize+1)/ssimShift)*sizeof(double));
    //for (int i=r1*r2*6+r2*6;i<r1*r2*6+r2*6+7;i++){
    ////for (int i=0;i<r1*r2*r3;i++){
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);
    //    printf("data%i=%e, %e\n",i, data1[i], data2[i]);

    //}
    int yNum = (8-ssimSize)/ssimShift+1;
    int yTotal = (r2-ssimSize)/ssimShift+1;
    int blksize = yTotal/yNum + (yTotal%yNum?1:0);

    const int csize = r3 * r2 * r1 * sizeof(float);

    hipMalloc((void**)&ddata1,   csize); 
    hipMalloc((void**)&ddata2,   csize); 
    hipMemcpy(ddata1,   data1,   csize, hipMemcpyHostToDevice); 
    hipMemcpy(ddata2,   data2,   csize, hipMemcpyHostToDevice); 

    const int rsize = r1 * r2*((r3-ssimSize+1)/ssimShift) * sizeof(double);

    hipMalloc((void**)&dresults, rsize); 
    hipMemcpy(dresults, results, rsize, hipMemcpyHostToDevice); 

    timer_GPU.StartCounter();
    dim3 dimBlock(32, 8);
    dim3 dimGrid(blksize, 1);
    type_three<<<dimGrid, dimBlock>>>(ddata1, ddata2, dresults, r3, r2, r1, ssimSize, ssimShift, yNum);

    hipMemcpy(results, dresults, rsize, hipMemcpyDeviceToHost); 
    double x=0, y=0;
    printf("GPU timing: %f ms\n", timer_GPU.GetCounter());
    for (int i=0; i<r1*r2*((r3-ssimSize+1)/ssimShift); i++){
        if (i%r1==0) {
printf("results%i=%e\n",i/r1-1,x);
x=0;

        }
        //if (i%r1==0) printf("delimiter%i\n",(i/r1));
        x += results[i];
        y += results[i];
        //printf("results%i=%e\n",i,results[i]);

    }
    printf("results=%e\n",y);

    hipFree(dresults);

    return 0;
}
