#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "cuZC_typeOne.h"
#include "matrix.hpp"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

__device__
void reduction(double sum1, double sum2,
        double minDiff, double maxDiff, double sumDiff, double sumOfDiffSquare, 
        double minErr, double maxErr, double sumErr, double sumErrSqr, double *results){

    //static __shared__ double shared[10*10];
    //dynamic shared mem
    extern __shared__ double shared[];

    int lane = threadIdx.x;
    int wid = threadIdx.y;


    for (int offset = warpSize/2; offset > 0; offset /= 2) 
    {
        minDiff = min(minDiff, __shfl_xor_sync(FULL_MASK, minDiff, offset));
        maxDiff = max(maxDiff, __shfl_xor_sync(FULL_MASK, maxDiff, offset));
        minErr = min(minErr, __shfl_xor_sync(FULL_MASK, minErr, offset));
        maxErr = max(maxErr, __shfl_xor_sync(FULL_MASK, maxErr, offset));
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
        sumDiff += __shfl_down_sync(FULL_MASK, sumDiff, offset);
        sumOfDiffSquare += __shfl_down_sync(FULL_MASK, sumOfDiffSquare, offset);
        sumErr += __shfl_down_sync(FULL_MASK, sumErr, offset);
        sumErrSqr += __shfl_down_sync(FULL_MASK, sumErrSqr, offset);
    }

    if (lane==0){
        shared[wid] = minDiff;
        shared[blockDim.y+wid] = maxDiff;
        shared[blockDim.y*2+wid] = minErr;
        shared[blockDim.y*3+wid] = maxErr;
        shared[blockDim.y*4+wid] = sum1;
        shared[blockDim.y*5+wid] = sum2;
        shared[blockDim.y*6+wid] = sumDiff;
        shared[blockDim.y*7+wid] = sumOfDiffSquare;
        shared[blockDim.y*8+wid] = sumErr;
        shared[blockDim.y*9+wid] = sumErrSqr;
    }

    __syncthreads();                  

    //if (wid==0)printf("ddata%i=%e:%e\n", 32*6+lane, shared[32*6+lane], ySum);

    if (wid==0){
        if (threadIdx.x < blockDim.y){
            minDiff = shared[lane];
            maxDiff = shared[blockDim.y+lane];
            minErr = shared[blockDim.y*2+lane];
            maxErr = shared[blockDim.y*3+lane];
            sum1 = shared[blockDim.y*4+lane];
            sum2 = shared[blockDim.y*5+lane];
            sumDiff = shared[blockDim.y*6+lane];
            sumOfDiffSquare = shared[blockDim.y*7+lane];
            sumErr = shared[blockDim.y*8+lane];
            sumErrSqr = shared[blockDim.y*9+lane];
        }else{
            minDiff = shared[0];  
            maxDiff = shared[blockDim.y]; 
            minErr = shared[blockDim.y*2]; 
            maxErr = shared[blockDim.y*3]; 
            sum1 = 0; 
            sum2 = 0;
            sumDiff = 0; 
            sumOfDiffSquare = 0;
            sumErr = 0;
            sumErrSqr = 0;
        }

        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            minDiff = min(minDiff, __shfl_xor_sync(FULL_MASK, minDiff, offset));
            maxDiff = max(maxDiff, __shfl_xor_sync(FULL_MASK, maxDiff, offset));
            minErr = min(minErr, __shfl_xor_sync(FULL_MASK, minErr, offset));
            maxErr = max(maxErr, __shfl_xor_sync(FULL_MASK, maxErr, offset));
            sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
            sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
            sumDiff += __shfl_down_sync(FULL_MASK, sumDiff, offset);
            sumOfDiffSquare += __shfl_down_sync(FULL_MASK, sumOfDiffSquare, offset);
            sumErr += __shfl_down_sync(FULL_MASK, sumErr, offset);
            sumErrSqr += __shfl_down_sync(FULL_MASK, sumErrSqr, offset);
        }
        
        if (lane==0){
            results[blockIdx.x] = minDiff;
            results[gridDim.x+blockIdx.x] = minErr;
            results[gridDim.x*2+blockIdx.x] = maxDiff;
            results[gridDim.x*3+blockIdx.x] = maxErr;
            results[gridDim.x*4+blockIdx.x] = sum1;
            results[gridDim.x*5+blockIdx.x] = sum2;
            results[gridDim.x*6+blockIdx.x] = sumDiff;
            results[gridDim.x*7+blockIdx.x] = sumOfDiffSquare;
            results[gridDim.x*8+blockIdx.x] = sumErr;
            results[gridDim.x*9+blockIdx.x] = sumErrSqr;
        }
    }
    //if(lane==0){
    //    if (sum1>0.0)printf("test%i,%i,%i:%e\n",lane,wid,blockIdx.x, sum1);

    //}

}

__device__
void gridReduction_cg(double *results) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    if (bid==0){
        double data = results[tidy*gridDim.x+tidx];

        for (int i=(tidx+blockDim.x); i<gridDim.x; i+=blockDim.x){
            if (tidy<2) data = min(data, results[tidy*gridDim.x+i]);
            else if (tidy<4) data = max(data, results[tidy*gridDim.x+i]);
            else data += results[tidy*gridDim.x+i];
        }
        __syncthreads();                  

        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            if (tidy<2) data = min(data, __shfl_xor_sync(FULL_MASK, data, offset));
            else if (tidy<4) data = max(data, __shfl_xor_sync(FULL_MASK, data, offset));
            else data += __shfl_down_sync(FULL_MASK, data, offset);
        }

        if (tidx==0) results[tidy*gridDim.x] = data;
    }
}

__global__ void type_one(float *data1, float *data2, double *diff, double *results, int r3, int r2, int r1, size_t ne) 
{
    cg::grid_group grid = cg::this_grid();

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float Data1, Data2;
    double Diff;
	double minDiff = data2[0]-data1[0];
	double maxDiff = minDiff;
	double sum1 = 0, sum2 = 0, sumDiff = 0, sumOfDiffSquare = 0; 
	
	double err;
	size_t numOfElem = ne;
	double minErr = fabs(minDiff);
	double maxErr = minErr;
	double sumErr = 0, sumErrSqr = 0;

    int i, j;

    for (j=tidy; j<r2; j+=blockDim.y){
        for (i=tidx; i<r1; i+=blockDim.x){
            Data1 = data1[bid*r1*r2+j*r1+i];
            Data2 = data2[bid*r1*r2+j*r1+i];
            sum1 += Data1; 
            sum2 += Data2;

            Diff = Data2 - Data1;
            diff[bid*r1*r2+j*r1+i] = Diff;
            minDiff = min(minDiff, Diff);
            maxDiff = max(maxDiff, Diff);
            sumDiff += Diff;
            sumOfDiffSquare += Diff * Diff;

            err = fabs(Diff);
            minErr = min(minErr, err);
            maxErr = max(maxErr, err);
            sumErr += err;
            sumErrSqr += err*err;
        }
    }
    __syncthreads();                  

    reduction(sum1, sum2, minDiff, maxDiff, sumDiff, sumOfDiffSquare, minErr, maxErr, sumErr, sumErrSqr, results);

    cg::sync(grid);

    gridReduction_cg(results);
//if (tid == 0)printf("ydata%i,%i=%e\n",Offsetx,Offsety, result);
//    if (tid==0) results[bid] = result;
    
}

__global__ void gridReduction(double *results, int r3) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    double data = results[tidy*r3+tidx];

    for (int i=(tidx+blockDim.x); i<r3; i+=blockDim.x){
        if (tidy<2) data = min(data, results[tidy*r3+i]);
        else if (tidy<4) data = max(data, results[tidy*r3+i]);
        else data += results[tidy*r3+i];
    }
    __syncthreads();                  

    for (int offset = warpSize/2; offset > 0; offset /= 2) 
    {
        if (tidy<2) data = min(data, __shfl_xor_sync(FULL_MASK, data, offset));
        else if (tidy<4) data = max(data, __shfl_xor_sync(FULL_MASK, data, offset));
        else data += __shfl_down_sync(FULL_MASK, data, offset);
    }

    if (tidx==0) results[tidy*r3] = data;
        
}
