#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "cuZC_ssim.h"
#include "cuZC_typeOne.h"
#include "matrix.hpp"

__device__
void reduction(double sum1, double sum2,
        double minDiff, double maxDiff, double sumDiff, double sumOfDiffSquare, 
        double minErr, double maxErr, double sumErr, double sumErrSqr, double *results){

    static __shared__ double shared[32*10];

    int lane = threadIdx.x;
    int wid = threadIdx.y;


    for (int offset = warpSize/2; offset > 0; offset /= 2) 
    {
        minDiff = min(minDiff, __shfl_xor_sync(FULL_MASK, minDiff, offset));
        maxDiff = max(maxDiff, __shfl_xor_sync(FULL_MASK, maxDiff, offset));
        minErr = min(minErr, __shfl_xor_sync(FULL_MASK, minErr, offset));
        maxErr = max(maxErr, __shfl_xor_sync(FULL_MASK, maxErr, offset));
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
        sumDiff += __shfl_down_sync(FULL_MASK, sumDiff, offset);
        sumOfDiffSquare += __shfl_down_sync(FULL_MASK, sumOfDiffSquare, offset);
        sumErr += __shfl_down_sync(FULL_MASK, sumErr, offset);
        sumErrSqr += __shfl_down_sync(FULL_MASK, sumErrSqr, offset);
    }

    if (lane==0){
        shared[wid] = minDiff;
        shared[32+wid] = maxDiff;
        shared[32*2+wid] = minErr;
        shared[32*3+wid] = maxErr;
        shared[32*4+wid] = sum1;
        shared[32*5+wid] = sum2;
        shared[32*6+wid] = sumDiff;
        shared[32*7+wid] = sumOfDiffSquare;
        shared[32*8+wid] = sumErr;
        shared[32*9+wid] = sumErrSqr;
    }

    __syncthreads();                  

    //if (wid==0)printf("ddata%i=%e:%e\n", 32*6+lane, shared[32*6+lane], ySum);

    if (wid==0){
        if (threadIdx.x < blockDim.y){
            minDiff = shared[lane];
            maxDiff = shared[32+lane];
            minErr = shared[32*2+lane];
            maxErr = shared[32*3+lane];
            sum1 = shared[32*4+lane];
            sum2 = shared[32*5+lane];
            sumDiff = shared[32*6+lane];
            sumOfDiffSquare = shared[32*7+lane];
            sumErr = shared[32*8+lane];
            sumErrSqr = shared[32*9+lane];
        }else{
            minDiff = shared[0];  
            maxDiff = shared[32]; 
            minErr = shared[32*2]; 
            maxErr = shared[32*3]; 
            sum1 = 0; 
            sum2 = 0;
            sumDiff = 0; 
            sumOfDiffSquare = 0;
            sumErr = 0;
            sumErrSqr = 0;
        }

        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            minDiff = min(minDiff, __shfl_xor_sync(FULL_MASK, minDiff, offset));
            maxDiff = max(maxDiff, __shfl_xor_sync(FULL_MASK, maxDiff, offset));
            minErr = min(minErr, __shfl_xor_sync(FULL_MASK, minErr, offset));
            maxErr = max(maxErr, __shfl_xor_sync(FULL_MASK, maxErr, offset));
            sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
            sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
            sumDiff += __shfl_down_sync(FULL_MASK, sumDiff, offset);
            sumOfDiffSquare += __shfl_down_sync(FULL_MASK, sumOfDiffSquare, offset);
            sumErr += __shfl_down_sync(FULL_MASK, sumErr, offset);
            sumErrSqr += __shfl_down_sync(FULL_MASK, sumErrSqr, offset);
        }
        
        if (lane==0){
            results[blockIdx.x] = minDiff;
            results[gridDim.x+blockIdx.x] = minErr;
            results[gridDim.x*2+blockIdx.x] = maxDiff;
            results[gridDim.x*3+blockIdx.x] = maxErr;
            results[gridDim.x*4+blockIdx.x] = sum1;
            results[gridDim.x*5+blockIdx.x] = sum2;
            results[gridDim.x*6+blockIdx.x] = sumDiff;
            results[gridDim.x*7+blockIdx.x] = sumOfDiffSquare;
            results[gridDim.x*8+blockIdx.x] = sumErr;
            results[gridDim.x*9+blockIdx.x] = sumErrSqr;
        }
    }
    //if(lane==0){
    //    if (sum1>0.0)printf("test%i,%i,%i:%e\n",lane,wid,blockIdx.x, sum1);

    //}

}

__global__ void type_one(float *data1, float *data2, double *diff, double *results, int r3, int r2, int r1, size_t ne) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float Data1, Data2;
    double Diff;
	double minDiff = data2[0]-data1[0];
	double maxDiff = minDiff;
	double sum1 = 0, sum2 = 0, sumDiff = 0, sumOfDiffSquare = 0; 
	
	double err;
	size_t numOfElem = ne;
	double minErr = fabs(minDiff);
	double maxErr = minErr;
	double sumErr = 0, sumErrSqr = 0;

    int i, j;

    for (j=tidy; j<r2; j+=blockDim.y){
        for (i=tidx; i<r1; i+=blockDim.x){
            Data1 = data1[bid*r1*r2+j*r1+i];
            Data2 = data2[bid*r1*r2+j*r1+i];
            sum1 += Data1; 
            sum2 += Data2;

            Diff = Data2 - Data1;
            diff[bid*r1*r2+j*r1+i] = Diff;
            minDiff = min(minDiff, Diff);
            maxDiff = max(maxDiff, Diff);
            sumDiff += Diff;
            sumOfDiffSquare += Diff * Diff;

            err = fabs(Diff);
            minErr = min(minErr, err);
            maxErr = max(maxErr, err);
            sumErr += err;
            sumErrSqr += err*err;
        }
    }
    __syncthreads();                  

    reduction(sum1, sum2, minDiff, maxDiff, sumDiff, sumOfDiffSquare, minErr, maxErr, sumErr, sumErrSqr, results);

//if (tid == 0)printf("ydata%i,%i=%e\n",Offsetx,Offsety, result);
//    if (tid==0) results[bid] = result;
    
}

__global__ void gridReduction(double *results, int r3) 
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    double data = results[tidy*r3+tidx];

    for (int i=(tidx+blockDim.x); i<r3; i+=blockDim.x){
        if (tidy<2) data = min(data, results[tidy*r3+i]);
        else if (tidy<4) data = max(data, results[tidy*r3+i]);
        else data += results[tidy*r3+i];
    }

    for (int offset = warpSize/2; offset > 0; offset /= 2) 
    {
        if (tidy<2) data = min(data, __shfl_xor_sync(FULL_MASK, data, offset));
        else if (tidy<4) data = max(data, __shfl_xor_sync(FULL_MASK, data, offset));
        else data += __shfl_down_sync(FULL_MASK, data, offset);
    }

    if (tidx==0) results[tidy*r3] = data;
        
}
