#include "hip/hip_runtime.h"
#include<cmath>
#include"cuZC_derivatives.h"

// __global__ void testKernel(float* data,int nx,int ny,int nz){
//     int n=nx*ny*nz;
//     float maxval=-inf;
//     for(int i=0;i<n;i++) maxval=max(maxval,data[i]);
//     printf("+++%.12f\n",maxval);
// }
//#ifndef float
//#define float float
//#endif

#ifndef arr3
#define arr3(x, y, z) ((x)*ny*nz+(y)*nz+(z))
#endif

#ifndef square
#define square(x) ((x)*(x))
#endif

#ifndef _BLOCK_SZ
#define BLOCKSZX 8
#define BLOCKSZY 8
#define BLOCKSZZ 8
#endif

#define blksz 1024
// must be 2^k, and 0<= k <= 10

#ifndef _INF
#define _INF
const float inf = 1. / 0.;
#endif


__global__ void maximumReduction(float *data, int p, int n) {

    int id = (blockIdx.x * blockDim.x + threadIdx.x) * p;

    if (id >= n) return;

    for (int stride = p; stride < p * blksz; stride <<= 1) {

        if (id % (stride << 1) == 0) {

            if (id + stride < n) {

                if (data[id + stride] > data[id]) {

                    data[id] = data[id + stride];
                }
            }
        }

        __syncthreads();
    }
}

float findMaximumUsingReduction(float *data, int n) {

    float *tem;

    hipMalloc(&tem, n * sizeof(float));

    hipMemcpy(tem, data, n * sizeof(float), hipMemcpyDeviceToDevice);

    int p = 1;

    while (true) {

        int blknum = n / (p * blksz) + (n % (p * blksz) > 0);

        maximumReduction<<<blknum, blksz>>>(tem, p, n);

        hipDeviceSynchronize();

        if (blknum == 1) break;

        p *= blksz;
    }

    float ans;

    hipMemcpy(&ans, tem, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(tem);

    return ans;
}

__global__ void minimumReduction(float *data, int p, int n) {

    int id = (blockIdx.x * blockDim.x + threadIdx.x) * p;

    if (id >= n) return;

    for (int stride = p; stride < p * blksz; stride <<= 1) {

        if (id % (stride << 1) == 0) {

            if (id + stride < n) {

                if (data[id + stride] < data[id]) {

                    data[id] = data[id + stride];
                }
            }
        }

        __syncthreads();
    }
}

float findMinimumUsingReduction(float *data, int n) {

    float *tem;

    hipMalloc(&tem, n * sizeof(float));

    hipMemcpy(tem, data, n * sizeof(float), hipMemcpyDeviceToDevice);

    int p = 1;

    while (true) {

        int blknum = n / (p * blksz) + (n % (p * blksz) > 0);

        minimumReduction<<<blknum, blksz>>>(tem, p, n);

        hipDeviceSynchronize();

        if (blknum == 1) break;

        p *= blksz;
    }

    float ans;

    hipMemcpy(&ans, tem, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(tem);

    return ans;
}

__global__ void sumReduction(float *data, int p, int n) {

    int id = (blockIdx.x * blockDim.x + threadIdx.x) * p;

    if (id >= n) return;

    for (int stride = p; stride < p * blksz; stride <<= 1) {

        if (id % (stride << 1) == 0) {

            if (id + stride < n) {

                data[id] += data[id + stride];
            }
        }

        __syncthreads();
    }
}

float sumupUsingReduction(float *data, int n) {

    float *tem;

    hipMalloc(&tem, n * sizeof(float));

    hipMemcpy(tem, data, n * sizeof(float), hipMemcpyDeviceToDevice);

    int p = 1;

    while (true) {

        int blknum = n / (p * blksz) + (n % (p * blksz) > 0);

        sumReduction<<<blknum, blksz>>>(tem, p, n);

        hipDeviceSynchronize();

        if (blknum == 1) break;

        p *= blksz;
    }

    float ans;

    hipMemcpy(&ans, tem, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(tem);

    return ans;
}

__global__ void sumReductionForceDouble(double *data, int p, int n) {

    int id = (blockIdx.x * blockDim.x + threadIdx.x) * p;

    if (id >= n) return;

    for (int stride = p; stride < p * blksz; stride <<= 1) {

        if (id % (stride << 1) == 0) {

            if (id + stride < n) {

                data[id] += data[id + stride];
            }
        }

        __syncthreads();
    }
}

double sumupUsingReductionForceDouble(double *data, int n) {

    double *tem;

    hipMalloc(&tem, n * sizeof(double));

    hipMemcpy(tem, data, n * sizeof(double), hipMemcpyDeviceToDevice);

    int p = 1;

    while (true) {

        int blknum = n / (p * blksz) + (n % (p * blksz) > 0);

        sumReductionForceDouble<<<blknum, blksz>>>(tem, p, n);

        hipDeviceSynchronize();

        if (blknum == 1) break;

        p *= blksz;
    }

    double ans;

    hipMemcpy(&ans, tem, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(tem);

    return ans;
}

__global__ void
derivativesKernel(float *data, float *dx, float *dy, float *dz, float *dx2, float *dy2, float *dz2, float *dxy, float *dyz, float *dzx, float *gl,
                  float *lap, int nx, int ny, int nz) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < nx && y < ny && z < nz) {

        // 1st order derivatives

        if (x < nx - 1) {
            dx[arr3(x, y, z)] = data[arr3(x + 1, y, z)] - data[arr3(x, y, z)];
        } else {
            dx[arr3(x, y, z)] = 0;
        }

        if (y < ny - 1) {
            dy[arr3(x, y, z)] = data[arr3(x, y + 1, z)] - data[arr3(x, y, z)];
        } else {
            dy[arr3(x, y, z)] = 0;
        }

        if (z < nz - 1) {
            dz[arr3(x, y, z)] = data[arr3(x, y, z + 1)] - data[arr3(x, y, z)];
        } else {
            dz[arr3(x, y, z)] = 0;
        }

        // 2rd order derivatives

        if (x < nx - 2) {
            dx2[arr3(x, y, z)] = data[arr3(x + 2, y, z)] - 2. * data[arr3(x + 1, y, z)] + data[arr3(x, y, z)];
        } else {
            dx2[arr3(x, y, z)] = 0;
        }

        if (y < ny - 2) {
            dy2[arr3(x, y, z)] = data[arr3(x, y + 2, z)] - 2. * data[arr3(x, y + 1, z)] + data[arr3(x, y, z)];
        } else {
            dy2[arr3(x, y, z)] = 0;
        }

        if (z < nz - 2) {
            dz2[arr3(x, y, z)] = data[arr3(x, y, z + 2)] - 2. * data[arr3(x, y, z + 1)] + data[arr3(x, y, z)];
        } else {
            dz2[arr3(x, y, z)] = 0;
        }

        if (x < nx - 1 && y < ny - 1) {

            dxy[arr3(x, y, z)] = data[arr3(x + 1, y + 1, z)] - data[arr3(x + 1, y, z)] - data[arr3(x, y + 1, z)] + data[arr3(x, y, z)];
        } else {

            dxy[arr3(x, y, z)] = 0;
        }

        if (y < ny - 1 && z < nz - 1) {

            dyz[arr3(x, y, z)] = data[arr3(x, y + 1, z + 1)] - data[arr3(x, y + 1, z)] - data[arr3(x, y, z + 1)] + data[arr3(x, y, z)];
        } else {

            dyz[arr3(x, y, z)] = 0;
        }

        if (z < nz - 1 && x < nx - 1) {

            dzx[arr3(x, y, z)] = data[arr3(x + 1, y, z + 1)] - data[arr3(x, y, z + 1)] - data[arr3(x + 1, y, z)] + data[arr3(x, y, z)];
        } else {

            dzx[arr3(x, y, z)] = 0;
        }

        //gradient length

        gl[arr3(x, y, z)] = sqrt(square(dx[arr3(x, y, z)]) + square(dy[arr3(x, y, z)]) + square(dz[arr3(x, y, z)]));

        //laplacian

        lap[arr3(x, y, z)] = square(dx2[arr3(x, y, z)]) + square(dy2[arr3(x, y, z)]) + square(dz2[arr3(x, y, z)]);
    }
}

__global__ void
squareMatrixKernelUsingBySobolev(float *data, float *dx, float *dy, float *dz, float *dx2, float *dy2, float *dz2, float *dxy, float *dyz, float *dzx,
                                 int nx, int ny, int nz) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < nx && y < ny && z < nz) {

        data[arr3(x, y, z)] *= data[arr3(x, y, z)];
        dx[arr3(x, y, z)] *= dx[arr3(x, y, z)];
        dy[arr3(x, y, z)] *= dy[arr3(x, y, z)];
        dz[arr3(x, y, z)] *= dz[arr3(x, y, z)];
        dx2[arr3(x, y, z)] *= dx2[arr3(x, y, z)];
        dy2[arr3(x, y, z)] *= dy2[arr3(x, y, z)];
        dz2[arr3(x, y, z)] *= dz2[arr3(x, y, z)];
        dxy[arr3(x, y, z)] *= dxy[arr3(x, y, z)];
        dyz[arr3(x, y, z)] *= dyz[arr3(x, y, z)];
        dzx[arr3(x, y, z)] *= dzx[arr3(x, y, z)];

    }
}

std::vector<float>
sobolev(float *data, float *dx, float *dy, float *dz, float *dx2, float *dy2, float *dz2, float *dxy, float *dyz, float *dzx, int nx, int ny,
        int nz) {

    dim3 blocksz = dim3(BLOCKSZX, BLOCKSZY, BLOCKSZZ);
    dim3 blocknum = dim3(nx / BLOCKSZX + (nx % BLOCKSZX > 0), ny / BLOCKSZY + (ny % BLOCKSZY > 0), nz / BLOCKSZZ + (nz % BLOCKSZZ > 0));

    squareMatrixKernelUsingBySobolev<<<blocknum, blocksz>>>(data, dx, dy, dz, dx2, dy2, dz2, dxy, dyz, dzx, nx, ny, nz);

    hipDeviceSynchronize();

    int n = nx * ny * nz;

    std::vector<float> s(3);

    s[0] = 0;
    s[0] += sumupUsingReduction(data, n);
    s[1] = s[0];
    s[1] += sumupUsingReduction(dx, n);
    s[1] += sumupUsingReduction(dy, n);
    s[1] += sumupUsingReduction(dz, n);
    s[2] = s[1];
    s[2] += sumupUsingReduction(dx2, n);
    s[2] += sumupUsingReduction(dy2, n);
    s[2] += sumupUsingReduction(dz2, n);
    s[2] += sumupUsingReduction(dxy, n);
    s[2] += sumupUsingReduction(dyz, n);
    s[2] += sumupUsingReduction(dzx, n);

    s[0] = sqrt(s[0] / n);
    s[1] = sqrt(s[1] / n);
    s[2] = sqrt(s[2] / n);

    return s;
}

__global__ void squareDifferenceKernelUsingByPSNR(float *f0, float *f1, double *tem, int nx, int ny, int nz) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < nx && y < ny && z < nz) {

        tem[arr3(x, y, z)] = f0[arr3(x, y, z)] - f1[arr3(x, y, z)];
        tem[arr3(x, y, z)] *= tem[arr3(x, y, z)];
    }
}

// __global__ void differenceKernelUsingByPSNR(float* f0,float* f1,float* tem,int nx,int ny,int nz){

//     int x=blockIdx.x*blockDim.x+threadIdx.x;
//     int y=blockIdx.y*blockDim.y+threadIdx.y;
//     int z=blockIdx.z*blockDim.z+threadIdx.z;

//     if(x<nx&&y<ny&&z<nz){

//         tem[arr3(x,y,z)]=abs(f0[arr3(x,y,z)]-f1[arr3(x,y,z)]);
//     }
// }

double findMSEn(float *f0, float *f1, int nx, int ny, int nz) {

    int n = nx * ny * nz;

    dim3 blocksz = dim3(BLOCKSZX, BLOCKSZY, BLOCKSZZ);
    dim3 blocknum = dim3(nx / BLOCKSZX + (nx % BLOCKSZX > 0), ny / BLOCKSZY + (ny % BLOCKSZY > 0), nz / BLOCKSZZ + (nz % BLOCKSZZ > 0));

    double *tem;

    hipMalloc(&tem, n * sizeof(double));

    squareDifferenceKernelUsingByPSNR<<<blocknum, blocksz>>>(f0, f1, tem, nx, ny, nz);

    hipDeviceSynchronize();

    double ans = sumupUsingReductionForceDouble(tem, n);

    hipFree(tem);

    return ans;
}

double findPSNR(float *f0, float *f1, int nx, int ny, int nz) {
//double findPSNR(float* f0,float* f1,int nx,int ny,int nz,int output_ae=0){

    int n = nx * ny * nz;

    double Rf0 = findMaximumUsingReduction(f0, n) - findMinimumUsingReduction(f0, n);

    if (Rf0 <= 0) return -inf;

    double MSEn = findMSEn(f0, f1, nx, ny, nz);

    if (MSEn <= 0) return inf;

    double PSNR = 20. * log10(Rf0 * sqrt(n) / sqrt(MSEn));

    // if(output_ae){

    //     float* tem;

    //     hipMalloc(&tem,n*sizeof(float));

    //     dim3 blocksz=dim3(BLOCKSZX,BLOCKSZY,BLOCKSZZ);
    //     dim3 blocknum=dim3(nx/BLOCKSZX+(nx%BLOCKSZX>0),ny/BLOCKSZY+(ny%BLOCKSZY>0),nz/BLOCKSZZ+(nz%BLOCKSZZ>0));
    //     differenceKernelUsingByPSNR<<<blocknum,blocksz>>>(f0,f1,tem,nx,ny,nz);

    //     float ae=findMaximumUsingReduction(tem,n);

    //     std::cout<<std::scientific<<"Absolute Error="<<ae<<std::endl;
    //     std::cout<<std::fixed;

    //     hipFree(tem);
    // }

    return PSNR;
}




// device -> device
void derivatives(float *data, int nx, int ny, int nz,
                 float *&dx, float *&dy, float *&dz,
                 float *&dx2, float *&dy2, float *&dz2,
                 float *&dxy, float *&dyz, float *&dzx,
                 float *&gl, float *&lap
) {

    int n = nx * ny * nz;

    hipMalloc(&dx, n * sizeof(float));
    hipMalloc(&dy, n * sizeof(float));
    hipMalloc(&dz, n * sizeof(float));
    hipMalloc(&dx2, n * sizeof(float));
    hipMalloc(&dy2, n * sizeof(float));
    hipMalloc(&dz2, n * sizeof(float));
    hipMalloc(&dxy, n * sizeof(float));
    hipMalloc(&dyz, n * sizeof(float));
    hipMalloc(&dzx, n * sizeof(float));
    hipMalloc(&gl, n * sizeof(float));
    hipMalloc(&lap, n * sizeof(float));

    dim3 blocksz = dim3(BLOCKSZX, BLOCKSZY, BLOCKSZZ);
    dim3 blocknum = dim3(nx / BLOCKSZX + (nx % BLOCKSZX > 0), ny / BLOCKSZY + (ny % BLOCKSZY > 0), nz / BLOCKSZZ + (nz % BLOCKSZZ > 0));

    derivativesKernel<<<blocknum, blocksz>>>(data, dx, dy, dz, dx2, dy2, dz2, dxy, dyz, dzx, gl, lap, nx, ny, nz);

    hipDeviceSynchronize();
}

// host -> device -> host
std::vector<float> derivativesPSNR(float *host_f0, float *host_f1, int nx, int ny, int nz) {

    int n = nx * ny * nz;

    float *f0;
    hipMalloc(&f0, n * sizeof(float));
    hipMemcpy(f0, host_f0, n * sizeof(float), hipMemcpyHostToDevice);

    float *f0_dx, *f0_dy, *f0_dz;
    float *f0_dx2, *f0_dy2, *f0_dz2;
    float *f0_dxy, *f0_dyz, *f0_dzx;
    float *f0_gl, *f0_lap;

    derivatives(f0, nx, ny, nz, f0_dx, f0_dy, f0_dz, f0_dx2, f0_dy2, f0_dz2, f0_dxy, f0_dyz, f0_dzx, f0_gl, f0_lap);
    std::vector<float> f0_sobolev = sobolev(f0, f0_dx, f0_dy, f0_dz, f0_dx2, f0_dy2, f0_dz2, f0_dxy, f0_dyz, f0_dzx, nx, ny, nz);

    float *f1;
    hipMalloc(&f1, n * sizeof(float));
    hipMemcpy(f1, host_f1, n * sizeof(float), hipMemcpyHostToDevice);

    float *f1_dx, *f1_dy, *f1_dz;
    float *f1_dx2, *f1_dy2, *f1_dz2;
    float *f1_dxy, *f1_dyz, *f1_dzx;
    float *f1_gl, *f1_lap;

    derivatives(f1, nx, ny, nz, f1_dx, f1_dy, f1_dz, f1_dx2, f1_dy2, f1_dz2, f1_dxy, f1_dyz, f1_dzx, f1_gl, f1_lap);
    std::vector<float> f1_sobolev = sobolev(f1, f1_dx, f1_dy, f1_dz, f1_dx2, f1_dy2, f1_dz2, f1_dxy, f1_dyz, f1_dzx, nx, ny, nz);

    std::vector<float> vec(12);

    vec[0] = findPSNR(f0, f1, nx, ny, nz);

    vec[1] = findPSNR(f0_dx, f1_dx, nx, ny, nz);
    vec[2] = findPSNR(f0_dy, f1_dy, nx, ny, nz);
    vec[3] = findPSNR(f0_dz, f1_dz, nx, ny, nz);

    vec[4] = findPSNR(f0_dx2, f1_dx2, nx, ny, nz);
    vec[5] = findPSNR(f0_dy2, f1_dy2, nx, ny, nz);
    vec[6] = findPSNR(f0_dz2, f1_dz2, nx, ny, nz);

    vec[7] = findPSNR(f0_gl, f1_gl, nx, ny, nz);
    vec[8] = findPSNR(f0_lap, f1_lap, nx, ny, nz);

    vec[9] = std::abs(f0_sobolev[0] - f1_sobolev[0]) / f0_sobolev[0];
    vec[10] = std::abs(f0_sobolev[1] - f1_sobolev[1]) / f0_sobolev[1];
    vec[11] = std::abs(f0_sobolev[2] - f1_sobolev[2]) / f0_sobolev[2];

    hipFree(f0);
    hipFree(f0_dx), hipFree(f0_dy), hipFree(f0_dz);
    hipFree(f0_dx2), hipFree(f0_dy2), hipFree(f0_dz2);
    hipFree(f0_dxy), hipFree(f0_dyz), hipFree(f0_dzx);
    hipFree(f0_gl), hipFree(f0_lap);

    hipFree(f1);
    hipFree(f1_dx), hipFree(f1_dy), hipFree(f1_dz);
    hipFree(f1_dx2), hipFree(f1_dy2), hipFree(f1_dz2);
    hipFree(f1_dxy), hipFree(f1_dyz), hipFree(f1_dzx);
    hipFree(f1_gl), hipFree(f1_lap);

    return vec;
}

#undef BLOCKSZX
#undef BLOCKSZY
#undef BLOCKSZZ
#undef blksz